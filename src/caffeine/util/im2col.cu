#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstring>

#include <hip/device_functions.h>

#include "caffeine/common.hpp"
#include "caffeine/util/im2col.hpp"

namespace caffeine {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
  const int height, const int width, const int ksize,
  const int stride, const int height_col, const int width_col, Dtype* data_col) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize * ksize;
    int h_in = h_out * stride;
    int w_in = w_out * stride;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize; ++i) {
      for (int j = 0; j < ksize; ++j) {
        *data_col = data_im[i * width + j];
        data_col += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int ksize, const int stride,
    Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height - ksize) / stride + 1;
  int width_col = (width - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  im2col_gpu_kernel<Dtype><<<CAFFEINE_GET_BLOCKS(num_kernels), CAFFEINE_CUDA_NUM_THREADS>>>(
    num_kernels, data_im, height, width, ksize, stride, height_col, width_col,
    data_col);
CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
  const int height, const int width, const int ksize, const int stride,
    float* data_col);
template void im2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int ksize, const int stride,
    double* data_col);


/*
// A bunch of stuff dealing with double atomic add
template <typename Dtype>
__device__ inline Dtype MyAtomicAdd(Dtype* address, Dtype val);

template <>
__device__ float MyAtomicAdd<float>(float* address, float val) {
  return atomicAdd(address, val);
}
template <>
__device__ double MyAtomicAdd<double>(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +
        __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
  const int height, const int width, const int ksize,
  const int stride, const int height_col, const int width_col, Dtype* data_im) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_out = index / height_col;
    int w_in = w_out * stride + channel_out % ksize;
    int h_in = h_out * stride + (channel_out / ksize) % ksize;
    int channel_in = channel_out / ksize / ksize;
    MyAtomicAdd(data_im + (channel_in * height + h_in) * width + w_in,
        data_col[(channel_out* height_col + h_out) * width_col + w_out]);
  }
}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int stride,
    Dtype* data_im) {
  CUDA_CHECK(hipMemset(data_im, 0, sizeof(Dtype) * height * width * channels));
  int height_col = (height - ksize) / stride + 1;
  int width_col = (width - ksize) / stride + 1;
  int channels_col = channels * ksize * ksize;
  int num_kernels = channels_col * height_col * width_col;
  col2im_gpu_kernel<Dtype><<<CAFFEINE_GET_BLOCKS(num_kernels), CAFFEINE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, ksize, stride, height_col, width_col,
      data_im);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int stride,
    float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int stride,
    double* data_im);
*/

}  // namespace caffeine
