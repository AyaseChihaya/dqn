#include "hip/hip_runtime.h"
#include "caffeine/layer.hpp"
#include "caffeine/vision_layers.hpp"
#include "caffeine/util/math_functions.hpp"

namespace caffeine {

template <typename Dtype>
__global__ void LRNFillScale(const int nthreads, const Dtype* in,
    const int num, const int channels, const int height,
    const int width, const int size, const Dtype alpha_over_size,
    Dtype* scale) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height;
    int offset = (n * channels * height + h) * width + w;
    int step = height * width;
    in += offset;
    scale += offset;
    int head = 0;
    int pre_pad = (size - 1) / 2;
    int post_pad = size - pre_pad - 1;
    Dtype accum_scale = 0;
    // fill the scale at [n, :, h, w]
    // accumulate values 
    while (head < post_pad) {
      accum_scale += in[head * step] * in[head * step];
      ++head;
    }
    // until we reach size, nothing needs to be subtracted
    while (head < size) {
      accum_scale += in[head * step] * in[head * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_scale += in[head * step] * in[head * step];
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
    // subtract only
    while (head < size + post_pad) {
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
  }
}

template <typename Dtype>
__global__ void LRNComputeOutput(const int nthreads, const Dtype* in,
    const Dtype* scale, const Dtype negative_beta, Dtype* out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    out[index] = in[index] * pow(scale[index], negative_beta);
  }
}

template <typename Dtype>
void LRNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  // First, compute scale
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  // We will launch one kernel for each pixel location, and have the kernel
  // go through all the channels.
  int n_threads = num_ * height_ * width_;
  LRNFillScale<<<CAFFEINE_GET_BLOCKS(n_threads), CAFFEINE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, num_, channels_, height_, width_, size_,
      alpha_ / size_, scale_data);
  CUDA_POST_KERNEL_CHECK;
  n_threads = bottom[0]->count();
  LRNComputeOutput<<<CAFFEINE_GET_BLOCKS(n_threads), CAFFEINE_CUDA_NUM_THREADS>>>(
      n_threads, bottom_data, scale_data, -beta_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
Dtype LRNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  NOT_IMPLEMENTED;
  return Dtype(0.);
}

INSTANTIATE_CLASS(LRNLayer);

}  // namespace caffeine
